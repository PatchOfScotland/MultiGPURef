

#include "lib/constants.cu.h"
#include "lib/helpers.cu.h"
#include "lib/map.cu"



typedef int funcType;

int main(){
    // Run times definitions
    float* runtimes_map_95 = (float*)malloc(sizeof(float)*ITERATIONS);
    float* runtimes_map_100 = (float*)malloc(sizeof(float)*ITERATIONS);
    float* runtimes_map_105 = (float*)malloc(sizeof(float)*ITERATIONS);

    float* runtimes_map_95_hinted = (float*)malloc(sizeof(float)*ITERATIONS);
    float* runtimes_map_100_hinted = (float*)malloc(sizeof(float)*ITERATIONS);
    float* runtimes_map_105_hinted = (float*)malloc(sizeof(float)*ITERATIONS);

    initHwd();
    size_t freeMem, totalMem;

    int Device;
    hipGetDevice(&Device);

    hipMemGetInfo(&freeMem, &totalMem);


    size_t capacity = freeMem / sizeof(funcType);

    // Map

    { // over Subscription: 0.95
        size_t arrSize = capacity / 2 * 0.95;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));

        init_array_cpu<funcType>(inputMem, 1337, arrSize);

        CUDA_RT_CALL(hipGetLastError());

        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_95, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    { // over Subscription: 1
        size_t arrSize = capacity / 2;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));

        init_array_cpu<funcType>(inputMem, 1337, arrSize);

        CUDA_RT_CALL(hipGetLastError());

        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_100, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    { // over Subscription: 1.05
        size_t arrSize = capacity / 2 * 1.05;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));

        init_array_cpu<funcType>(inputMem, 1337, arrSize);

        CUDA_RT_CALL(hipGetLastError());

        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_105, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    // Map

    { // over Subscription Hinted: 0.95
        size_t arrSize = capacity / 2 * 0.95;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));



        init_array_cpu<funcType>(inputMem, 1337, arrSize);
        CUDA_RT_CALL(hipMemAdvise(inputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemAdvise(outputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemPrefetchAsync(inputMem, bufferSize, Device));
        CUDA_RT_CALL(hipDeviceSynchronize());


        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_95_hinted, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    { // over Subscription Hinted: 1
        size_t arrSize = capacity / 2;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));

        init_array_cpu<funcType>(inputMem, 1337, arrSize);
        CUDA_RT_CALL(hipMemAdvise(inputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemAdvise(outputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemPrefetchAsync(inputMem, bufferSize, Device));
        CUDA_RT_CALL(hipDeviceSynchronize());

        CUDA_RT_CALL(hipGetLastError());

        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_100_hinted, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    { // over Subscription hinted: 1.05
        size_t arrSize = capacity / 2 * 1.05;
        size_t bufferSize = arrSize*sizeof(funcType);

        funcType* inputMem;
        funcType* outputMem;
        CUDA_RT_CALL(hipMallocManaged(&inputMem, bufferSize));
        CUDA_RT_CALL(hipMallocManaged(&outputMem, bufferSize));

        init_array_cpu<funcType>(inputMem, 1337, arrSize);
        CUDA_RT_CALL(hipMemAdvise(inputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemAdvise(outputMem, bufferSize, hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemPrefetchAsync(inputMem, bufferSize, Device));
        CUDA_RT_CALL(hipDeviceSynchronize());

        CUDA_RT_CALL(hipGetLastError());

        void* args[] = { &inputMem, &outputMem, &arrSize };
        hipError_t (*function)(void**) = &singleGPU::ApplyMapVoidArgs< MapP2 <funcType>>;

        benchmarkFunction(function, args, runtimes_map_105_hinted, ITERATIONS);

        CUDA_RT_CALL(hipFree(inputMem));
        CUDA_RT_CALL(hipFree(outputMem));
    }

    for(int i = 0; i < ITERATIONS; i++){
            std::cout << runtimes_map_95[i] << ", " << runtimes_map_100[i] << ", " << runtimes_map_105[i] << ", ";
            std::cout << runtimes_map_95_hinted[i] << ", " << runtimes_map_100_hinted[i] << ", " << runtimes_map_105_hinted[i] << "\n";
        }


    //Freeing Stuff


}