#include "hip/hip_runtime.h"
#include "constants.cu.h"
#include "map.cu"
#include "helpers.cu.h"

#include <iostream>
#include <fstream>

#define ARRAY_SIZE 1e6
#define ITERATIONS 25

#define LOGGING 1

typedef float funcType;

template<class T>
class MapP2 {
    public:
        typedef T InpElTp;
        typedef T RedElTp;

        static __device__ __host__ RedElTp apply(const InpElTp i) {return i+2;};
};

template<class T>
class MapBasic {
    public:
        typedef T InpElTp;
        typedef T RedElTp;

        static __device__ __host__ RedElTp apply(const InpElTp i) {return i * i ;};
};

int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }

    #if LOGGING
    std::ofstream logging;
    logging.open("HWINFO.log");
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    logging << "Number of devices: " << deviceCount << "\n";
    for (int i = 0; i < deviceCount; i++){
        hipDeviceProp_t properties;
        hipGetDeviceProperties(&properties, i);
        logging << "Device " << i << " name: " << properties.name << "\n";
        logging << "Device can use Unified Memory:" << properties.unifiedAddressing << "\n";
    }
    for (int i = 0; i < deviceCount; i++){
        for(int j = 0; j < deviceCount; j++){
            if (i==j) continue;
            int canAccessPeer = 0;
            hipDeviceCanAccessPeer(&canAccessPeer, i,j);
            if (canAccessPeer){
                logging << "Device "<< i << " can access Device " << j << "\n";
            } else {
                logging << "Device "<< i << " cannot access Device " << j << "\n";
            }
        }
    }


    #endif

    size_t N = ARRAY_SIZE;
    size_t data_size = N * sizeof(float);

    funcType* d_in; 
    funcType* d_out_multiGPU;
    funcType* d_out_singleGPU;

    gpuAssert(hipMallocManaged((void**)&d_in, data_size));
    gpuAssert(hipMallocManaged((void**)&d_out_singleGPU, data_size));
    gpuAssert(hipMallocManaged((void**)&d_out_multiGPU, data_size));

    init_arr< funcType >(d_in, 1337, N);


    singleGPU::ApplyMap< MapBasic<funcType> >(d_in, d_out_singleGPU, N);
    multiGPU::ApplyMap< MapBasic<funcType> >(d_in, d_out_multiGPU, N);

    if (!compare_arrays_nummeric<funcType>(d_out_singleGPU, d_out_multiGPU, N)){
        output << "INVALID RESULTS!";
    } else {
        for(int i = 0; i < ITERATIONS; i++ ){
            hipEvent_t start_event_m, stop_event_m;
            hipEvent_t start_event_s, stop_event_s;

            gpuAssert(hipEventCreate(&start_event_s));
            gpuAssert(hipEventCreate(&stop_event_s));
            gpuAssert(hipEventCreate(&start_event_m));
            gpuAssert(hipEventCreate(&stop_event_m));

            gpuAssert(hipEventRecord(start_event_s));
            gpuAssert(singleGPU::ApplyMap< MapBasic<funcType> >(d_in, d_out_singleGPU, N));
            gpuAssert(hipEventRecord(stop_event_s));
            gpuAssert(hipEventSynchronize(stop_event_s));

            gpuAssert(hipEventRecord(start_event_m));
            gpuAssert(multiGPU::ApplyMap< MapBasic<funcType> >(d_in, d_out_multiGPU, N));
            gpuAssert(hipEventRecord(stop_event_m));
            gpuAssert(hipEventSynchronize(stop_event_m));

            float ms_s = 0;
            float ms_m = 0;
            gpuAssert(hipEventElapsedTime(&ms_s, start_event_s, stop_event_s));
            gpuAssert(hipEventElapsedTime(&ms_m, start_event_m, stop_event_m));
            output << ms_s << ", " << ms_m << "\n";

            gpuAssert(hipEventDestroy(start_event_s));
            gpuAssert(hipEventDestroy(stop_event_s));
            gpuAssert(hipEventDestroy(start_event_m));
            gpuAssert(hipEventDestroy(stop_event_m));
        }
    }


    output.close();

    hipFree(d_in);
    hipFree(d_out_multiGPU);
    hipFree(d_out_singleGPU);

    return 0;
}

