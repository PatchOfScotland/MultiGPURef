#include "hip/hip_runtime.h"
#ifndef STENCIL_H
#define STENCIL_H

#include "helpers.cu.h"
#include "constants.cu.h"
#include "scan.cu"

#define TOL 1e-6
#define MAX_ITER 1000
#define X 4096
#define Y 4096


__global__ void init_boundaries(float* __restrict__ const a1, const float pi, const int h, const int w ){
    int64_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < h) {
        const float y_value = sin(2.0 * pi * idx / (h - 1));
        a1[idx * w] = y_value;
        a1[idx * w + (w - 1)] = y_value;
    }
    
}

hipError_t init_stencil(float* __restrict__ const a, const int h, const int w){
    const int threads = 1024;
    hipMemset(a, 0, h * w * sizeof(float));
    size_t numblocks = (h + threads - 1 ) / threads;
    init_boundaries<<<numblocks, threads>>>(a, CR_CUDART_PI, h, w);
    return hipGetLastError();
}

namespace singleGPU{

    __global__ void jacobiKernel (
            float* src, 
            float* dst, 
            float* l2_norm,
            const int h, 
            const int w
        ) {
        //Pull this into shared memory with boarders
        extern __shared__ char shmem[];
        float* scanMem   = (float*) shmem;
        float* FocusArea = (float*) shmem;

        const int16_t FocusAreaSize = (blockDim.y + 2)*(blockDim.x + 2);
        const int16_t flat_idx = blockDim.x * threadIdx.y + threadIdx.x;

        for(int16_t flat_local_Idx = flat_idx; 
            flat_local_Idx < FocusAreaSize; 
            flat_local_Idx += blockDim.x * blockDim.y){
            const int16_t Focus_area_x = flat_local_Idx % (blockDim.x + 2);
            const int16_t Focus_area_y = flat_local_Idx / (blockDim.x + 2);

            const int64_t RealIdx_x = blockIdx.x * blockDim.x + Focus_area_x - 1;
            const int64_t RealIdx_y = blockIdx.y * blockDim.y + Focus_area_y - 1;

            bool in_border_x = 0 <= RealIdx_x && RealIdx_x < w;
            bool in_border_y = 0 <= RealIdx_y && RealIdx_y < h;

            if (in_border_x && in_border_y) {
                FocusArea[Focus_area_y * blockDim.x + Focus_area_x] = src[RealIdx_y * w + RealIdx_x];
            } else {
                FocusArea[Focus_area_y * blockDim.x + Focus_area_x] = 0;
            }
        }
        const int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t y = blockIdx.y * blockDim.y + threadIdx.y;



        if(x < w && y < h){

            __syncthreads();

        const float new_value = (FocusArea[(threadIdx.y + 1) * blockDim.x + threadIdx.x + 2] + 
                                 FocusArea[(threadIdx.y + 1) * blockDim.x + threadIdx.x] + 
                                 FocusArea[(threadIdx.y + 2) * blockDim.x + threadIdx.x + 1] + 
                                 FocusArea[threadIdx.y * blockDim.x + threadIdx.x + 1]) / 4;


        dst[y*w + x] = new_value;
        const float local_norm = powf(new_value - src[y * w + x], 2);   

        scanMem[flat_idx] = local_norm;
        __syncthreads();
        scanIncBlock< Add <float> >(scanMem, flat_idx);
        __syncthreads();
        }
        if(flat_idx == 0){
            atomicAdd(l2_norm, scanMem[ blockDim.x * blockDim.y - 1]);
        }
    }

    hipError_t jacobi(float* src, float* dst, const int h, const int w){
        int iter   = 0;
        float norm = 1.0;

        float* norm_d;

        CUDA_RT_CALL(hipMallocManaged(&norm_d, sizeof(float) ));

        const int blockSize = 32;


        const int rowBlocks = (h % blockSize == 0) ? h / blockSize : h / blockSize + 1;
        const int colBlocks = (w % blockSize == 0) ? w / blockSize : w / blockSize + 1; 
        // MemAdvices
        
        const size_t shmemSize = (blockSize + 2) * (blockSize + 2) * sizeof(float);
        const dim3 block(blockSize, blockSize, 1);

        while(norm > TOL && iter < MAX_ITER){
            hipMemset(norm_d, 0, sizeof(float));

            dim3 grid(colBlocks, rowBlocks, 1);
            jacobiKernel<<<grid, block, shmemSize>>>(
                src, 
                dst, 
                norm_d, 
                h, 
                w
            );
            
            DeviceSyncronize();
            
            norm = std::sqrt(*norm_d);
            std::swap(src, dst);
            iter++;
        }
        std::cout << "Jacobi completed with " << iter << " iterations and Norm " << norm << "\n";
        return hipGetLastError();
    }
}

namespace multiGPU {
    __global__ void jacobiKernel (
            float* src, 
            float* dst, 
            float* l2_norm,
            const int h, 
            const int w, 
            const int devID
        ) {
        //Pull this into shared memory with boarders
        extern __shared__ char shmem[];
        float* scanMem   = (float*) shmem;
        float* FocusArea = (float*) shmem;

        const int16_t FocusAreaSize = (blockDim.y + 2)*(blockDim.x + 2);
        const int16_t flat_idx = blockDim.x * threadIdx.y + threadIdx.x;

        for(int16_t flat_local_Idx = flat_idx; 
            flat_local_Idx < FocusAreaSize; 
            flat_local_Idx += blockDim.x * blockDim.y){
            const int16_t Focus_area_x = flat_local_Idx % (blockDim.x + 2);
            const int16_t Focus_area_y = flat_local_Idx / (blockDim.x + 2);

            const int64_t RealIdx_x = blockIdx.x * blockDim.x + Focus_area_x - 1;
            const int64_t RealIdx_y = blockDim.y * gridDim.y * devID + blockIdx.y * blockDim.y + Focus_area_y - 1;

            bool in_border_x = 0 <= RealIdx_x && RealIdx_x < w;
            bool in_border_y = 0 <= RealIdx_y && RealIdx_y < h;

            if (in_border_x && in_border_y) {
                FocusArea[Focus_area_y * blockDim.x + Focus_area_x] = src[RealIdx_y * w + RealIdx_x];
            } else {
                FocusArea[Focus_area_y * blockDim.x + Focus_area_x] = 0;
            }
        }
        const int64_t x = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t y = blockDim.y * gridDim.y * devID + blockIdx.y * blockDim.y + threadIdx.y;



        if(x < w && y < h){

            __syncthreads();

        const float new_value = (FocusArea[(threadIdx.y + 1) * blockDim.x + threadIdx.x + 2] + 
                                 FocusArea[(threadIdx.y + 1) * blockDim.x + threadIdx.x] + 
                                 FocusArea[(threadIdx.y + 2) * blockDim.x + threadIdx.x + 1] + 
                                 FocusArea[threadIdx.y * blockDim.x + threadIdx.x + 1]) / 4;


        dst[y*w + x] = new_value;
        const float local_norm = powf(new_value - src[y * w + x], 2);   

        scanMem[flat_idx] = local_norm;
        __syncthreads();
        scanIncBlock< Add <float> >(scanMem, flat_idx);
        __syncthreads();
        }
        if(flat_idx == 0){
            atomicAdd(l2_norm, scanMem[ blockDim.x * blockDim.y - 1]);
        }
    }



    hipError_t jacobi(float* src, float* dst, const int h, const int w){

        int Device;
        hipGetDevice(&Device);
        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);

        int iter   = 0;
        float norm = 1.0;

        float* norm_d;

        CUDA_RT_CALL(hipMallocManaged(&norm_d, DeviceCount*sizeof(float) ));

        const int blockSize = 32;


        const int rowBlocks = (h % blockSize == 0) ? h / blockSize : h / blockSize + 1;
        const int colBlocks = (w % blockSize == 0) ? w / blockSize : w / blockSize + 1; 
        // MemAdvices
        int rows_per_device_low  = rowBlocks / DeviceCount;
        int rows_per_device_high = rows_per_device_low + 1;
        int highRows = rowBlocks % DeviceCount;

        int offset_rows = 0;
        for(int devID = 0; devID < DeviceCount; devID++){
            size_t brows = (devID < highRows) ? rows_per_device_high : rows_per_device_low;
            size_t elems = brows*w*blockSize;
            CUDA_RT_CALL(hipMemAdvise(src + offset_rows*blockSize*w, elems*sizeof(float), hipMemAdviseSetPreferredLocation, devID));        
            //Rows above and below
            if(devID != 0){
                CUDA_RT_CALL(hipMemAdvise(src + (offset_rows-1)*blockSize*w, w*sizeof(float), hipMemAdviseSetAccessedBy, devID));
            }
            if(devID != DeviceCount -1){
                CUDA_RT_CALL(hipMemAdvise(src + (offset_rows + brows)*blockSize*w, w*sizeof(float), hipMemAdviseSetAccessedBy, devID));
            }

            CUDA_RT_CALL(hipMemAdvise(dst + offset_rows*blockSize*w, elems*sizeof(float), hipMemAdviseSetPreferredLocation, devID));        
            //Rows above and below
            if(devID != 0){
                CUDA_RT_CALL(hipMemAdvise(dst + (offset_rows-1)*blockSize*w, w*sizeof(float), hipMemAdviseSetAccessedBy, devID));
            }
            if(devID != DeviceCount -1){
                CUDA_RT_CALL(hipMemAdvise(dst + (offset_rows + brows)*blockSize*w, w*sizeof(float), hipMemAdviseSetAccessedBy, devID));
            }
            offset_rows += brows;
        }

        

        const size_t shmemSize = (blockSize + 2) * (blockSize + 2) * sizeof(float);
        const dim3 block(blockSize, blockSize, 1);

        while(norm > TOL && iter < MAX_ITER){
            hipMemset(norm_d,0, DeviceCount*sizeof(float));

            for(int devID = 0; devID < DeviceCount; devID++){
                hipSetDevice(devID);
                size_t brows = (devID < highRows) ? rows_per_device_high : rows_per_device_low;
                dim3 grid(colBlocks, brows,1);
                jacobiKernel<<<grid, block, shmemSize>>>(
                    src, 
                    dst, 
                    norm_d + devID, 
                    h, 
                    w, 
                    devID
                );
            }
            DeviceSyncronize();
            norm = 0.0;
            for(int devID=0;devID < DeviceCount; devID++){
                norm += norm_d[devID];
            }
            norm = std::sqrt(norm);
            std::swap(src, dst);
            iter++;
        }
        std::cout << "Jacobi completed with " << iter << " iterations and Norm " << norm << "\n";
        return hipGetLastError();
    }
}


#endif