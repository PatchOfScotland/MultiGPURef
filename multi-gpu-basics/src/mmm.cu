#ifndef MMM_H
#define MMM_H

    
namespace singleGPU {

template <class ElTp, int T> 
__global__ void matMultRegTiledKernel(
        ElTp* A,
        ElTp* B,
        ElTp* C, 
        int heightA, 
        int widthB, 
        int widthA
    ) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  bidy * T;
        //int const bdimx = blockDim.x; // =Tile
        //int const bdimy = blockDim.y; // =Tile

        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + bidy * T < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
            __syncthreads();


            for(int i = 0; i < T; i++){
                if ((ii + i) < heightA && j < widthB)  {
                    C[(i + ii)*widthB + j] = Creg[i];
                }
            }
        }
    }



    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {
            dim3 block(T, T, 1);
            int grid_x = ceil((float)B_width / (T * T));
            int grid_y = ceil((float)A_height / (T)); 
            dim3 grid(grid_x, grid_y, 1);


            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);
            return hipGetLastError();
    }
}    

namespace multiGPU {

    template <class ElTp, int T> 
    __global__ void matMultRegTiledKernel(ElTp* A, ElTp* B, ElTp* C, int heightA, int widthB, int widthA, int deviceCount) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  gridDim.y * deviceCount + bidy * T;


        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy +   gridDim.y * deviceCount + bidy * T < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
            __syncthreads();
            for(int i = 0; i < T; i++){
                if ((ii + i) < heightA && j < widthB)  {
                    C[(i + ii)*widthB + j] = Creg[i];
                }
            }
        }
    }


    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {

        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);


        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + DeviceCount - 1) / DeviceCount; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);

        for(int dev_id = 0; dev_id < DeviceCount; dev_id++){
            hipSetDevice(dev_id);
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        return hipGetLastError();
    }

    

}

#endif