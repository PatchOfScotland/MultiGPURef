#ifndef MMM_H
#define MMM_H
#include <iostream> // for debuggin
    
#define HEIGHT_A 16384   
#define HEIGHT_B 16384  // Given that HEIGHT_B = WIDTH_A
#define WIDTH_B  16384



namespace singleGPU {

    template <class ElTp, int T> 
    __global__ void matMultRegTiledKernel(
        ElTp* A,
        ElTp* B,
        ElTp* C, 
        int heightA, 
        int widthB, 
        int widthA
    ) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  bidy * T;
        //int const bdimx = blockDim.x; // =Tile
        //int const bdimy = blockDim.y; // =Tile

        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + bidy * T < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
            __syncthreads();


            for(int i = 0; i < T; i++){
                if ((ii + i) < heightA && j < widthB)  {
                    C[(i + ii)*widthB + j] = Creg[i];
                }
            }
        }
    }



    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {
            dim3 block(T, T, 1);
            int grid_x = ceil((float)B_width / (T * T));
            int grid_y = ceil((float)A_height / (T)); 
            dim3 grid(grid_x, grid_y, 1);


            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);
            return hipGetLastError();
    }

    template<class ElTp, int T>
    __global__ void matMultTrivial(ElTp* A, ElTp* B, ElTp* C, int A_height, int B_width, int B_height){
        const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t j = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (i < B_width || j < A_height) {
            int accum = 0;
            for(int k = 0; k < B_height; k++){
                accum += A[j*B_height + k] * B[k*B_width + i];
            }
            C[j * A_height + i] = accum;
        }
    }

    template< class ElTp, int T>
    hipError_t MMM_trivial(
        ElTp* A,
        ElTp* B, 
        ElTp* C, 
        int A_height, 
        int B_width, 
        int B_height
    ) {
        dim3 block(T, T, 1);
        int grid_x = ceil((float)B_width / (T));
        int grid_y = ceil((float)A_height / (T)); 
        dim3 grid(grid_x, grid_y, 1);

        matMultTrivial< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);


        return hipPeekAtLastError();
    }


}    

namespace multiGPU {

    template <class ElTp, int T> 
    __global__ void matMultRegTiledKernel(ElTp* A, ElTp* B, ElTp* C, int heightA, int widthB, int widthA, int devID) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA; 
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  gridDim.y * T * devID + bidy * T;


        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + ii < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b; 
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
        }
        __syncthreads();
        for(int i = 0; i < T; i++){
            if ((ii + i) < heightA && j < widthB)  {
                C[(i + ii) * widthB + j] = Creg[i];
            }
        }
    }


    template< class ElTp, int T>
    hipError_t MMM(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height
        ) {

        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);

        int Device = -1;
        hipGetDevice(&Device);

        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + DeviceCount - 1) / DeviceCount; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);

        for(int dev_id = 0; dev_id < DeviceCount; dev_id++){
            hipSetDevice(dev_id);
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        
        hipSetDevice(Device);

        return hipGetLastError();
    }

    template< class ElTp, int T>
    hipError_t MMM_emulated(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height,
            int emulatedDevices
        ) {
        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + emulatedDevices - 1) / emulatedDevices; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);


        for(int dev_id = 0; dev_id < emulatedDevices; dev_id++){
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        return hipGetLastError();
    }       

    template<class ElTp, int T>
    __global__ void matMultTrivial(ElTp* A, ElTp* B, ElTp* C, int A_height, int B_width, int B_height, int devID){
        const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
        const int64_t j = devID * gridDim.y * blockDim.y  + blockIdx.y * blockDim.y + threadIdx.y;
        
        if (i < B_width || j < A_height) {
            int accum = 0;
            for(int k = 0; k < B_height; k++){
                accum += A[j*B_height + k] * B[k*B_width + i];
            }
            C[j * A_height + i] = accum;
        }
    }

    template< class ElTp, int T>
    hipError_t MMM_trivial_emulated(
            ElTp* A,
            ElTp* B, 
            ElTp* C, 
            int A_height, 
            int B_width, 
            int B_height,
            int emulatedDevices
        ) {
        dim3 block(T, T, 1);
        //std::cout << A_height << ", " << B_width << ", " << B_height << ", " << T <<  "\n";

        int grid_x_total = ceil((float)B_width / (T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + emulatedDevices - 1) / emulatedDevices; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);


        for(int dev_id = 0; dev_id < emulatedDevices; dev_id++){
            matMultTrivial< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }
        return hipGetLastError();
    } 
    
    template< class ElTp, int T>
    hipError_t MMM_adviced_prefetch(
        ElTp* A,
        ElTp* B, 
        ElTp* C, 
        int A_height, 
        int B_width, 
        int B_height
    ){
        int Device = -1;
        hipGetDevice(&Device);
        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);

        size_t A_size = A_height * B_height * sizeof(ElTp);
        size_t B_size = B_width  * B_height * sizeof(ElTp);

        dim3 block(T, T, 1);
        int grid_x_total = ceil((float)B_width / (T * T));
        int grid_y_total = ceil((float)A_height / (T)); 
        
        int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        int grid_y = (grid_y_total + DeviceCount - 1) / DeviceCount; // Same trick to get matching blocksizes

        dim3 grid(grid_x, grid_y, 1);
        size_t grid_byte_count = grid_x* grid_y * T * T * sizeof(ElTp);
        
        
        //cudaStream_t deviceStream[DeviceCount];

        for(int devID = 0; devID < DeviceCount; devID++){
            //cudaStreamCreate(&deviceStream[devID]);
            cudaMemAdvise(A, A_size, hipMemAdviseSetReadMostly, devID);
            cudaMemAdvise(B, B_size, hipMemAdviseSetReadMostly, devID);
            
            cudaMemPrefetchAsync(A, A_size, devID);
            cudaMemPrefetchAsync(B, B_size, devID);

            size_t offset = devID * grid_byte_count;
            cudaMemAdvise(C + offset, grid_byte_count, hipMemAdviseSetAccessedBy, devID);
            cudaMemAdvise(C + offset, grid_byte_count, hipMemAdviseSetPreferredLocation, devID);

        }
        
        //cudaMemAdvise()


        for(int devID = 0; devID < DeviceCount; devID++){
            hipSetDevice(devID);
            matMultRegTiledKernel< ElTp, T ><<<grid, block >>>(A,B,C, A_height, B_width, B_height, devID);

        }
        
        //for(int devID = 0; devID < DeviceCount; devID++){
        //    cudaStreamDestroy(deviceStream[devID]);
        //}

        hipSetDevice(Device);

        return hipGetLastError();
    }


}

#endif