#include <iostream>
#include <fstream>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <sstream>
#include <cstdlib>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "scan.cu"


#define DEFAULT_N 5e8
#define DEFAULT_OUTPUTFILE "data/scan_bench.csv"

typedef int funcType;

template <typename T>
T get_argval(char** begin, char** end, const std::string& arg, const T default_val) {
    T argval = default_val;
    char** itr = std::find(begin, end, arg);
    if (itr != end && ++itr != end) {
        std::istringstream inbuf(*itr);
        inbuf >> argval;
    }
    return argval;
}

bool get_arg(char** begin, char** end, const std::string& arg) {
    char** itr = std::find(begin, end, arg);
    if (itr != end) {
        return true;
    }
    return false;
}

int main(int argc, char* argv[]) {
    const int64_t N = get_argval<int>(argv, argv + argc, "-x", DEFAULT_N);
    const std::string outputFile = get_argval<std::string>(argv, argv + argc, "-output", DEFAULT_OUTPUTFILE);
 
    std::ofstream File(outputFile);

    initHwd();
    EnablePeerAccess();

    int DeviceCount;
    hipGetDeviceCount(&DeviceCount);

    funcType* data_in;
    funcType* data_out_single;
    funcType* data_out_multi_device;
    funcType* data_tmp;
    funcType* data_tmp_multi_device;

    hipMallocManaged(&data_in, N*sizeof(funcType));
    hipMallocManaged(&data_out_single, N*sizeof(funcType));
    hipMallocManaged(&data_out_multi_device, N*sizeof(funcType));
    hipMallocManaged(&data_tmp, MAX_BLOCK*sizeof(funcType));
    hipMallocManaged(&data_tmp_multi_device, MAX_BLOCK*sizeof(funcType));

    funcType* device_data_in;
    funcType* device_data_out;
    funcType* aggregates;
    funcType* inc_prefix;
    uint8_t*  flags;
    
    hipEvent_t syncEvent[DeviceCount];
    hipEvent_t scan1blockEvent;

    hipEventCreateWithFlags(&scan1blockEvent, hipEventDisableTiming);

    for(int devID = 0; devID < DeviceCount; devID++){
      hipSetDevice(devID);
      hipEventCreateWithFlags(&syncEvent[devID], hipEventDisableTiming);
    }


    hipMalloc(&device_data_in, N*sizeof(funcType));
    hipMalloc(&device_data_out, N*sizeof(funcType));
    AllocateFlagArray<Add<funcType> >(&flags, &aggregates, &inc_prefix, N);

    init_array_cpu<funcType>(data_in, 1337, N);
    hipMemcpy(device_data_in, data_in, N*sizeof(funcType), hipMemcpyDefault);

    DeviceSyncronize();    

    for(int run = 0; run < ITERATIONS + 1; run++){
        float ms_single, ms_2pass, ms_MD;
        
        hipEvent_t start_single;
        hipEvent_t stop_single;

        CUDA_RT_CALL(hipEventCreate(&start_single));
        CUDA_RT_CALL(hipEventCreate(&stop_single));

        CUDA_RT_CALL(hipEventRecord(start_single));
        scanInc< Add < funcType > >(1024, N, data_out_single, data_in, data_tmp);
        CUDA_RT_CALL(hipEventRecord(stop_single));
        DeviceSyncronize();
        CUDA_RT_CALL(hipEventElapsedTime(&ms_single, start_single, stop_single));

        hipEventDestroy(start_single);
        hipEventDestroy(stop_single);

        hipEvent_t start_2pass;
        hipEvent_t stop_2pass;

        CUDA_RT_CALL(hipEventCreate(&start_2pass));
        CUDA_RT_CALL(hipEventCreate(&stop_2pass));

        CUDA_RT_CALL(hipEventRecord(start_2pass));
        scanWrapper< Add <funcType> >(device_data_out, device_data_in, N, flags, aggregates, inc_prefix);
        CUDA_RT_CALL(hipEventRecord(stop_2pass));
        DeviceSyncronize();
        CUDA_RT_CALL(hipEventElapsedTime(&ms_2pass, start_2pass, stop_2pass));

        hipEventDestroy(start_2pass);
        hipEventDestroy(stop_2pass);

        hipEvent_t start_MD;
        hipEvent_t stop_MD;

        CUDA_RT_CALL(hipEventCreate(&start_MD));
        CUDA_RT_CALL(hipEventCreate(&stop_MD));

        CUDA_RT_CALL(hipEventRecord(start_MD));
        scanInc_multiDevice< Add < funcType > >(1024, N, data_out_single, data_in, data_tmp, syncEvent, scan1blockEvent);
        CUDA_RT_CALL(hipEventRecord(stop_MD));
        DeviceSyncronize();
        CUDA_RT_CALL(hipEventElapsedTime(&ms_MD, start_MD, stop_MD));

        hipEventDestroy(start_MD);
        hipEventDestroy(stop_MD);

        if(run != 0) File << ms_single << ", " << ms_2pass << ", " << ms_MD << "\n";
    }

    File.close();

    hipFree(data_in);
    hipFree(data_out_single);
    hipFree(data_out_multi_device);
    hipFree(data_tmp);
    hipFree(data_tmp_multi_device);
    hipFree(device_data_in);
    hipFree(aggregates);
    hipFree(inc_prefix);
    hipFree(flags);


    return 0;
}