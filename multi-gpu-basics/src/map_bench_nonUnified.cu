#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "map.cu"



#define ENABLEPEERACCESS 1

typedef int funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    int Device; 
    hipGetDevice(&Device);
    int Devices;
    hipGetDeviceCount(&Devices);
    hipStream_t streams[Devices];

    for(int devID = 0; devID < Devices; devID++){
        hipSetDevice(devID);
        hipStreamCreate(&streams[devID]);
    }

    hipError_t e;
    funcType* h_in;
    funcType* d_in[Devices];
    funcType* d_out[Devices];

    size_t dataSize = ARRAY_LENGTH*sizeof(funcType);

    h_in = (funcType*)malloc(dataSize);

    for(int devID = 0; devID < Devices; devID++){
        CUDA_RT_CALL(hipMallocManaged(&d_in[devID], ARRAY_LENGTH*sizeof(funcType)));
        CUDA_RT_CALL(hipMallocManaged(&d_out[devID], ARRAY_LENGTH*sizeof(funcType)));
    }
    
    init_array_cpu< funcType >(h_in, 1337, ARRAY_LENGTH);
    

    for(int run = 0; run < ITERATIONS + 1; run++){
        hipEvent_t start_event, stop_event;

        CUDA_RT_CALL(hipEventCreate(&start_event));
        CUDA_RT_CALL(hipEventCreate(&stop_event));

        CUDA_RT_CALL(hipEventRecord(start_event));
        e = multiGPU::ApplyMapNonUnified< MapBasic<funcType> >(h_in, d_in, d_out, ARRAY_LENGTH);
        CUDA_RT_CALL(e);
        CUDA_RT_CALL(hipDeviceSynchronize());
        CUDA_RT_CALL(hipEventRecord(stop_event));
        CUDA_RT_CALL(hipEventSynchronize(stop_event));

        float ms;
        CUDA_RT_CALL(hipEventElapsedTime(&ms, start_event, stop_event));
        if(run != 0) output << ms << "\n";
    }

    free(h_in);
    for(int devID = 0; devID < Devices; devID++){
        hipFree(d_in[devID]);
        hipFree(d_out[devID]);    
    }

    
}