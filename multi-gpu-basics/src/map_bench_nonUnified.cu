#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "map.cu"



#define ENABLEPEERACCESS 1

typedef int funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    int Device; 
    hipGetDevice(&Device);
    int Devices;
    hipGetDeviceCount(&Devices);
    hipStream_t streams[Devices];

    for(int devID = 0; devID < Devices; devID++){
        hipSetDevice(devID);
        hipStreamCreate(&streams[devID]);
    }

    hipError_t e;
    funcType* h_in;
    funcType* d_in[Devices];
    funcType* d_out[Devices];

    size_t dataSize = ARRAY_LENGTH*sizeof(funcType);

    h_in = (funcType*)malloc(dataSize);
    init_array_cpu< funcType >(h_in, 1337, ARRAY_LENGTH);

    int ArrayPerDevice = (ARRAY_LENGTH + Devices - 1) / Devices;

    for(int devID = 0; devID < Devices; devID++){
        int offset = ArrayPerDevice * devID;
        CUDA_RT_CALL(hipMalloc(&d_in[devID], ArrayPerDevice*sizeof(funcType)));
        CUDA_RT_CALL(hipMalloc(&d_out[devID], ArrayPerDevice*sizeof(funcType)));
        CUDA_RT_CALL(hipMemcpy(d_in[devID], h_in + offset, ArrayPerDevice*sizeof(funcType), hipMemcpyDefault));
    }
    
    

    for(int run = 0; run < ITERATIONS + 1; run++){
        hipEvent_t start_event, stop_event;

        CUDA_RT_CALL(hipEventCreate(&start_event));
        CUDA_RT_CALL(hipEventCreate(&stop_event));

        CUDA_RT_CALL(hipEventRecord(start_event));
        e = multiGPU::ApplyMapNonUnified< MapBasic<funcType> >(d_in, d_out, ARRAY_LENGTH);
        CUDA_RT_CALL(e);
        syncronize();
        CUDA_RT_CALL(hipEventRecord(stop_event));
        CUDA_RT_CALL(hipEventSynchronize(stop_event));

        float ms;
        CUDA_RT_CALL(hipEventElapsedTime(&ms, start_event, stop_event));
        if(run != 0) output << ms << "\n";
    }

    free(h_in);
    for(int devID = 0; devID < Devices; devID++){
        hipFree(d_in[devID]);
        hipFree(d_out[devID]);    
    }

    
}