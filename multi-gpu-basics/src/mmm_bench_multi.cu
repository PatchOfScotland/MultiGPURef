#include <iostream>
#include <chrono>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "mmm.cu"

#define HEIGHT_A 1024   
#define HEIGHT_B 1024  // Given that HEIGHT_B = WIDTH_A
#define WIDTH_B  1024

#define TILE 16

#define ENABLEPEERACCESS 1

typedef int funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    size_t A_length = HEIGHT_A * HEIGHT_B;
    size_t B_length = HEIGHT_B * WIDTH_B;
    size_t C_length = HEIGHT_A * WIDTH_B;

    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    int Device = -1;
    hipGetDevice(&Device);

    funcType* A;
    funcType* B;
    funcType* C_multi;

    
    CUDA_RT_CALL(hipMallocManaged(&A,        A_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&B,        B_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&C_multi,  C_length*sizeof (funcType)));
    // may make this multicore?    
    CUDA_RT_CALL(init_arr< funcType >(A, 1337, A_length));
    CUDA_RT_CALL(init_arr< funcType >(B, 420, B_length));
    hipDeviceSynchronize();

    for(int run = 0; run < ITERATIONS; run++){
        hipEvent_t start_event, stop_event;

        CUDA_RT_CALL(hipEventCreate(&start_event));
        CUDA_RT_CALL(hipEventCreate(&stop_event));

        CUDA_RT_CALL(hipEventRecord(start_event));
        hipError_t e = multiGPU::MMM< funcType, TILE >(A,B,C_multi, HEIGHT_A, WIDTH_B, HEIGHT_B);
        CUDA_RT_CALL(e);
        hipSetDevice(Device);
        CUDA_RT_CALL(hipEventRecord(stop_event));
        CUDA_RT_CALL(hipEventSynchronize(stop_event));

        float ms;
        CUDA_RT_CALL(hipEventElapsedTime(&ms, start_event, stop_event));
        if(run != 0) output << ms << "\n";
    }

    hipFree(A);
    hipFree(B);
    hipFree(C_multi);
}