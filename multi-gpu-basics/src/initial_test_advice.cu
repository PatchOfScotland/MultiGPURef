#include <iostream>
#include <chrono>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"

#define N 1e7


#define ENABLEPEERACCESS 1

typedef int funcType;

int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif


    funcType* A;
    float ms;
    hipEvent_t start, stop;
    int Device = -1;
    hipGetDevice(&Device);

    for(int run = 0; run < ITERATIONS + 1; run++){
        
        gpuAssert(hipMallocManaged(&A, N*sizeof(funcType)));
        
        gpuAssert(hipMemAdvise(&A, N*sizeof(funcType), hipMemAdviseSetPreferredLocation, Device));

        gpuAssert(hipEventRecord(start));
        gpuAssert(init_arr< funcType >(A, 1337, N));
        gpuAssert(hipEventRecord(stop));
        gpuAssert(hipEventSynchronize(stop));

        gpuAssert(hipEventElapsedTime(&ms, start, stop));
        output << ms << "\n";

        hipFree(A);

    }
    
    // may make this multicore?    


}