#include "helpers.cu.h"
#include "constants.cu.h"

#define HEIGHT 32
#define WIDTH  32

#define TILE 16


int main(int argc, char* argv[]){
    
    int* data;

    hipMallocManaged(&data, HEIGHT*WIDTH * sizeof(int));


    multiGPU::iotaMatrix_emulate<int, TILE>(data, HEIGHT, WIDTH, 3);

    hipDeviceSynchronize();
    printMatrix<int>(data, HEIGHT, WIDTH);

    return 0;
}