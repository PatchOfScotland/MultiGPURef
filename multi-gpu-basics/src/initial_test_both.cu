#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"

#define N 1e6


#define ENABLEPEERACCESS 1

typedef int funcType;

int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif
    
    funcType* A;
    float ms;
    hipEvent_t start, stop;
    int Device = -1;
    hipGetDevice(&Device);

    for(int run = 0; run < ITERATIONS; run++){
        CUDA_RT_CALL(hipEventCreate(&start));
        CUDA_RT_CALL(hipEventCreate(&stop));

        CUDA_RT_CALL(hipEventRecord(start));
        CUDA_RT_CALL(hipMallocManaged(&A, N*sizeof(funcType)));

        CUDA_RT_CALL(hipMemAdvise(A, N*sizeof(funcType), hipMemAdviseSetPreferredLocation, Device));
        CUDA_RT_CALL(hipMemAdvise(A, N*sizeof(funcType), hipMemAdviseSetAccessedBy, Device));
        CUDA_RT_CALL(hipMemPrefetchAsync(A, N*sizeof(funcType), Device));

        CUDA_RT_CALL(init_arr< funcType >(A, 1337, N));
        CUDA_RT_CALL(hipEventRecord(stop));
        CUDA_RT_CALL(hipEventSynchronize(stop));

        CUDA_RT_CALL(hipEventElapsedTime(&ms, start, stop));
        output << ms << "\n";

        CUDA_RT_CALL(hipEventDestroy(start));
        CUDA_RT_CALL(hipEventDestroy(stop));

        hipFree(A);
        hipDeviceReset();

    }
    
    // may make this multicore?    


}