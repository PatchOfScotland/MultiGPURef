#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "atomic.cu"

#define THREADSSIZE 1e7


int main(int argc, const char** argv){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    EnablePeerAccess(); 
    
    int* single_atomic_address;
    int* single_atomic_system_address;
    int* multi_atomic_address;
    int* multi_atomic_system_address;

    hipMallocManaged(&single_atomic_address, sizeof(int));
    hipMallocManaged(&multi_atomic_address, sizeof(int));
    hipMallocManaged(&single_atomic_system_address, sizeof(int));
    hipMallocManaged(&multi_atomic_system_address, sizeof(int));

    for(int run = 0; run < ITERATIONS + 1; run++){

        hipEvent_t single_atomic_start;
        hipEvent_t single_system_atomic_start;
        hipEvent_t multi_atomic_start;
        hipEvent_t multi_system_atomic_start;

        hipEvent_t single_atomic_stop;
        hipEvent_t single_system_atomic_stop;
        hipEvent_t multi_atomic_stop;
        hipEvent_t multi_system_atomic_stop;

        hipEventCreate(&single_atomic_start);
        hipEventCreate(&single_system_atomic_start);
        hipEventCreate(&multi_atomic_start);
        hipEventCreate(&multi_system_atomic_start);

        hipEventCreate(&single_atomic_stop);
        hipEventCreate(&single_system_atomic_stop);
        hipEventCreate(&multi_atomic_stop);
        hipEventCreate(&multi_system_atomic_stop);

        hipEventRecord(single_atomic_start);
        singleGPU::atomicTest(single_atomic_address, THREADSSIZE);
        DeviceSyncronize();
        hipEventRecord(single_atomic_stop);
        
        hipEventRecord(single_system_atomic_start);
        singleGPU::atomicSystemTest(single_atomic_system_address, THREADSSIZE);
        DeviceSyncronize();
        hipEventRecord(single_system_atomic_stop);

        hipEventRecord(multi_atomic_start);
        singleGPU::atomicTest(multi_atomic_address, THREADSSIZE);
        DeviceSyncronize();
        hipEventRecord(multi_atomic_stop);

        hipEventRecord(multi_system_atomic_start);
        singleGPU::atomicTest(multi_atomic_system_address, THREADSSIZE);
        DeviceSyncronize();
        hipEventRecord(multi_system_atomic_stop);

        float ms_single;
        float ms_single_system;
        float ms_multi;
        float ms_multi_system;

        hipEventElapsedTime(&ms_single, single_atomic_start, single_atomic_stop);
        hipEventElapsedTime(&ms_single_system, single_system_atomic_start, single_system_atomic_stop);
        hipEventElapsedTime(&ms_multi, multi_atomic_start, multi_atomic_stop);
        hipEventElapsedTime(&ms_multi_system, multi_system_atomic_start, multi_system_atomic_stop);

        output << ms_single << ", " << ms_single_system << ", " << ms_multi << ", " << ms_multi_system << "\n";

        hipEventDestroy(single_atomic_start);
        hipEventDestroy(single_system_atomic_start);
        hipEventDestroy(multi_atomic_start);
        hipEventDestroy(multi_system_atomic_start);

        hipEventDestroy(single_atomic_stop);
        hipEventDestroy(single_system_atomic_stop);
        hipEventDestroy(multi_atomic_stop);
        hipEventDestroy(multi_system_atomic_stop);


    }


}