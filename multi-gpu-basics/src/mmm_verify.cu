#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "mmm.cu"

#define HEIGHT_A 256    
#define HEIGHT_B 256 // Given that HEIGHT_B = WIDTH_A
#define WIDTH_B  256

#define TILE 16

#define ENABLEPEERACCESS 1

typedef int funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    size_t A_length = HEIGHT_A * HEIGHT_B;
    size_t B_length = HEIGHT_B * WIDTH_B;
    size_t C_length = HEIGHT_A * WIDTH_B;

    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    hipError_t e;

    funcType* A;
    funcType* B;
    funcType* C_single;
    funcType* C_multi;
    funcType* C_trivial;


    CUDA_RT_CALL(hipMallocManaged(&A,        A_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&B,        B_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&C_single, C_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&C_multi,  C_length*sizeof (funcType)));
    CUDA_RT_CALL(hipMallocManaged(&C_trivial,  C_length*sizeof (funcType)));

    CUDA_RT_CALL(init_arr< funcType >(A, 1337, A_length));
    CUDA_RT_CALL(init_arr< funcType >(B, 420, B_length));
    hipDeviceSynchronize();

    e = singleGPU::MMM< funcType, TILE >(A,B,C_single, HEIGHT_A, WIDTH_B, HEIGHT_B);
    CUDA_RT_CALL(e);
    e = multiGPU::MMM_trivial_emulated< funcType, TILE >(A,B,C_trivial,HEIGHT_A, WIDTH_B, HEIGHT_B, 3);
    CUDA_RT_CALL(e);
    e = multiGPU::MMM_emulated< funcType, TILE >(A, B,C_multi, HEIGHT_A, WIDTH_B, HEIGHT_B,3);
    CUDA_RT_CALL(e);

    hipDeviceSynchronize();

    if(compare_arrays<funcType>(C_single, C_trivial, C_length)){
        std::cout << "Trivial is correct\n";
    } else {
        std::cout << "Trivial is incorrect\n";
        printMatrix<funcType>(A, HEIGHT_A, HEIGHT_B);
        std::cout << "\n\n";
        printMatrix<funcType>(B, HEIGHT_B, WIDTH_B);
        std::cout << "\n\n";
        printMatrix<funcType>(C_trivial, HEIGHT_A, WIDTH_B);
        std::cout << "\n\n";
        printMatrix<funcType>(C_single, HEIGHT_A, WIDTH_B);
        std::cout << "\n\n";
    }

    
    if(compare_arrays< funcType >(C_single, C_multi, C_length)){
        std::cout << "Valid output\n";
    } else {
        std::cout << "Invalid Result \n";
        for(int i = 0; i < C_length; i++){
            if (abs(C_single[i] - C_multi[i]) > EPSILON){
                //std::cout << C_single[i] << " " << C_multi[i] << " " << i << "\n";
            }
        }
    }


    hipFree(A);
    hipFree(B);
    hipFree(C_single);
    hipFree(C_multi);

}