#include "hip/hip_runtime.h"
#ifndef MAP_H
#define MAP_H

#include "helpers.cu.h"
#include "constants.cu.h"

#define ARRAY_LENGTH 1e9

template<class T>
class MapP2 {
    public:
        typedef T InpElTp;
        typedef T RedElTp;

        static __device__ __host__ RedElTp apply(const InpElTp i) {return i+2;};
};

template<class T>
class MapBasic {
    public:
        typedef T InpElTp;
        typedef T RedElTp;

        static __device__ __host__ RedElTp apply(const InpElTp i) {return i * i ;};
};




namespace multiGPU {
    template<class MapFunc>
    __global__ void MapMultiGPU(typename MapFunc::InpElTp* input, typename MapFunc::RedElTp* output, int deviceID, size_t N){
        int64_t idx = deviceID * gridDim.x*blockDim.x + blockDim.x*blockIdx.x + threadIdx.x;
        if(idx < N){
            output[idx] = MapFunc::apply(input[idx]);
        }
    }


    template<class MapFunc>
    hipError_t ApplyMap(
        typename MapFunc::InpElTp* input,
        typename MapFunc::RedElTp* output,
        size_t N
    ){
        int DeviceNum;
        hipGetDeviceCount(&DeviceNum);

            size_t allocated_per_device = (N + DeviceNum - 1) / DeviceNum; 
        size_t num_blocks           = (allocated_per_device + BLOCKSIZE - 1 ) / BLOCKSIZE;
    
        for(int devID=0; devID < DeviceNum; devID++){
            hipSetDevice(devID);
            MapMultiGPU< MapFunc ><<<num_blocks, BLOCKSIZE>>>(input, output, devID, N);
        }
        hipSetDevice(0);
        return hipGetLastError();
    }

    template<class MapFunc>
    hipError_t ApplyMapPrefetchAdvice(
        typename MapFunc::InpElTp* input,
        typename MapFunc::RedElTp* output,
        size_t N
    ){
        int DeviceNum;
        hipGetDeviceCount(&DeviceNum);

        size_t allocated_per_device = (N + DeviceNum - 1) / DeviceNum;
        size_t dataSize = allocated_per_device*sizeof(typename MapFunc::InpElTp);
        size_t num_blocks = (allocated_per_device + BLOCKSIZE - 1 ) / BLOCKSIZE;
        for(int devID = 0; devID < DeviceNum; devID++){
            int offset = devID * allocated_per_device;
            hipMemAdvise(input + offset, dataSize, hipMemAdviseSetReadMostly, devID);
            hipMemPrefetchAsync(input + offset, dataSize, devID);
            hipMemAdvise(output + offset, dataSize, hipMemAdviseSetAccessedBy, devID);
            hipMemAdvise(output + offset, dataSize, hipMemAdviseSetPreferredLocation, devID);
        }

        for(int devID=0; devID < DeviceNum; devID++){
            hipSetDevice(devID);
            MapMultiGPU< MapFunc ><<<num_blocks, BLOCKSIZE>>>(input, output, devID, N);
        }
        hipSetDevice(0);
        return hipGetLastError();
    }

    template<class MapFunc>
    hipError_t ApplyMapNonUnified(
        typename MapFunc::InpElTp* h_input,
        typename MapFunc::InpElTp* d_input[],
        typename MapFunc::RedElTp* output[],
        size_t N
    ){
        int DeviceNum;
        hipGetDeviceCount(&DeviceNum);

        size_t allocated_per_device = (N + DeviceNum - 1) / DeviceNum;
        size_t dataSize =  allocated_per_device*sizeof(typename MapFunc::InpElTp);
        size_t num_blocks = (allocated_per_device + BLOCKSIZE - 1 ) / BLOCKSIZE;
        for(int devID = 0; devID < DeviceNum; devID++){
            int offset = devID * allocated_per_device;
            hipMemcpy(d_input[devID], h_input + offset, dataSize, hipMemcpyHostToDevice);
        }

        for(int devID=0; devID < DeviceNum; devID++){
            hipSetDevice(devID);
            MapMultiGPU< MapFunc ><<<num_blocks, BLOCKSIZE>>>(d_input[devID], output[devID], devID, N);
        }
        hipSetDevice(0);
        return hipGetLastError();
    }

    template<class MapFunc>
    hipError_t ApplyMapStreams(
        typename MapFunc::InpElTp* input,
        typename MapFunc::RedElTp* output,
        size_t N,
        hipStream_t streams[]
    ){

        int DeviceNum;
        hipGetDeviceCount(&DeviceNum);

        size_t allocated_per_device = (N + DeviceNum - 1) / DeviceNum;
        size_t dataSize =  allocated_per_device*sizeof(typename MapFunc::InpElTp);
        size_t num_blocks           = (allocated_per_device + BLOCKSIZE - 1 ) / BLOCKSIZE;
        for(int devID = 0; devID < DeviceNum; devID++){
            int offset = devID * allocated_per_device;
            hipMemAdvise(input + offset, dataSize, hipMemAdviseSetReadMostly, devID);
            hipMemPrefetchAsync(input + offset, dataSize, devID, streams[devID]);
            hipMemAdvise(output + offset, dataSize, hipMemAdviseSetAccessedBy, devID);
            hipMemAdvise(output + offset, dataSize, hipMemAdviseSetPreferredLocation, devID);
        }

        for(int devID=0; devID < DeviceNum; devID++){
            MapMultiGPU< MapFunc ><<<num_blocks, BLOCKSIZE, 0, streams[devID]>>>(input, output, devID, N);
        }
        return hipGetLastError();
    }
}


namespace singleGPU {

    template<class MapFunc>
    __global__ void MapGPU(
        typename MapFunc::InpElTp* input,
        typename MapFunc::RedElTp* output,
        size_t N
    ){
            int64_t idx = blockDim.x*blockIdx.x + threadIdx.x;
            if (idx < N) {
                output[idx] = MapFunc::apply(input[idx]);
            }
        }


    template<class MapFunc>
    hipError_t ApplyMap(
        typename MapFunc::InpElTp* input,
        typename MapFunc::RedElTp* output,
        size_t N
    ){
        size_t num_blocks = (N + BLOCKSIZE - 1 ) / BLOCKSIZE;
        MapGPU< MapFunc ><<<num_blocks, BLOCKSIZE >>>(input, output, N);
        return hipGetLastError();
    }
}


#endif