#include <iostream>
#include <chrono>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "mmm.cu"

#define HEIGHT_A 2048
#define HEIGHT_B 2048 // Given that HEIGHT_B = WIDTH_A
#define WIDTH_B  2048

#define TILE 16

typedef float funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    size_t A_length = HEIGHT_A * HEIGHT_B;
    size_t B_length = HEIGHT_B * WIDTH_B;
    size_t C_length = HEIGHT_A * WIDTH_B;

    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif


    funcType* A;
    funcType* B;
    funcType* C_single;
    funcType* C_multi;


    gpuAssert(hipMallocManaged(&A,        A_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&B,        B_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&C_single, C_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&C_multi,  C_length*sizeof (funcType)));

    gpuAssert(init_arr< funcType >(A, 1337, A_length));
    gpuAssert(init_arr< funcType >(B, 420, B_length));
    hipDeviceSynchronize();

    gpuAssert(singleGPU::MMM< funcType, TILE >(A,B,C_single, HEIGHT_A, WIDTH_B, HEIGHT_B));
    gpuAssert(multiGPU::MMM< funcType, TILE >(A,B,C_multi, HEIGHT_A, WIDTH_B, HEIGHT_B));
    hipDeviceSynchronize();

    
    if(compare_arrays< funcType >(C_single, C_multi, C_length)){
        output << "Valid output\n";
        
    } else {
        output << "Invalid Result \n";
    }

    for(int run = 0; run < ITERATIONS; run++){


        //hipEvent_t start_event_m, stop_event_m;
        //hipEvent_t start_event_s, stop_event_s;

        //gpuAssert(hipEventCreate(&start_event_s));
        //gpuAssert(hipEventCreate(&stop_event_s));
        //gpuAssert(hipEventCreate(&start_event_m));
        //gpuAssert(hipEventCreate(&stop_event_m));

        //gpuAssert(hipEventRecord(start_event_s));
        auto start_single = std::chrono::high_resolution_clock::now();
        gpuAssert(singleGPU::MMM< funcType, TILE >(A,B,C_single, HEIGHT_A, WIDTH_B, HEIGHT_B));
        hipDeviceSynchronize();
        auto stop_single = std::chrono::high_resolution_clock::now();
        //gpuAssert(hipEventRecord(stop_event_s));
        //gpuAssert(hipEventSynchronize(stop_event_s));

        //gpuAssert(hipEventRecord(start_event_m));
        auto start_multi = std::chrono::high_resolution_clock::now();
        gpuAssert(multiGPU::MMM< funcType, TILE >(A,B,C_multi, HEIGHT_A, WIDTH_B, HEIGHT_B));
        hipDeviceSynchronize();
        auto stop_multi = std::chrono::high_resolution_clock::now();
        //gpuAssert(hipEventRecord(stop_event_m));
        //gpuAssert(hipEventSynchronize(stop_event_m));

        auto ms_s = std::chrono::duration_cast<std::chrono::microseconds>(stop_single - start_single);
        auto ms_m = std::chrono::duration_cast<std::chrono::microseconds>(stop_multi - start_multi);
        //gpuAssert(hipEventElapsedTime(&ms_s, start_event_s, stop_event_s));
        //gpuAssert(hipEventElapsedTime(&ms_m, start_event_m, stop_event_m));
        output << ms_s.count() << ", " << ms_m.count() << "\n";

        //gpuAssert(hipEventDestroy(start_event_s));
        //gpuAssert(hipEventDestroy(stop_event_s));
        //gpuAssert(hipEventDestroy(start_event_m));
        //gpuAssert(hipEventDestroy(stop_event_m));
    }

    hipFree(A);
    hipFree(B);
    hipFree(C_single);
    hipFree(C_multi);

}