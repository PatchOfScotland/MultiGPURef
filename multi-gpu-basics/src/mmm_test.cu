#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "mmm.cu"

#define HEIGHT_A 1024
#define HEIGHT_B 1024 // Given that HEIGHT_B = WIDTH_A
#define WIDTH_B  1024

#define TILE 16

typedef float funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    size_t A_length = HEIGHT_A * HEIGHT_B;
    size_t B_length = HEIGHT_B * WIDTH_B;
    size_t C_length = HEIGHT_A * WIDTH_B;

    funcType* A;
    funcType* B;
    funcType* C_single;
    funcType* C_multi;


    gpuAssert(hipMallocManaged(&A,        A_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&B,        B_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&C_single, C_length*sizeof(funcType)));
    gpuAssert(hipMallocManaged(&C_multi,  C_length*sizeof (funcType)));
    
    gpuAssert(init_arr< funcType >(A, 1337, A_length));
    gpuAssert(init_arr< funcType >(B, 420, B_length));

    gpuAssert(singleGPU::MMM< funcType, TILE >(A,B,C_single, HEIGHT_A, WIDTH_B, HEIGHT_B));
    gpuAssert(multiGPU::MMM< funcType, TILE >(A,B,C_multi, HEIGHT_A, WIDTH_B, HEIGHT_B));

    if(compare_arrays_nummeric< funcType >(C_single, C_multi, C_length)){
        output << "valid Result \n";
    } else {
        output << "Invalid Result \n";
    }

    hipFree(A);
    hipFree(B);
    hipFree(C_single);
    hipFree(C_multi);

}