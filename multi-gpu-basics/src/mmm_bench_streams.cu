#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "mmm.cu"


#define TILE 16

#define ENABLEPEERACCESS 1

typedef int funcType;

int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
        
    size_t A_length = HEIGHT_A * HEIGHT_B;
    size_t B_length = HEIGHT_B * WIDTH_B;
    size_t C_length = HEIGHT_A * WIDTH_B;

    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    int Device; 
    hipGetDevice(&Device);
    int Devices;
    hipGetDeviceCount(&Devices);
    hipStream_t streams[Devices];

    for(int devID = 0; devID < Devices; devID++){
        hipSetDevice(devID);
        hipStreamCreate(&streams[devID]);
    }
    hipSetDevice(Device);

    hipError_t e;
    funcType* A;
    funcType* B;
    funcType* C;


    CUDA_RT_CALL(hipMallocManaged(&A, A_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&B, B_length*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&C, C_length*sizeof(funcType)));

    init_array_cpu< funcType >(A, 1337, A_length);
    init_array_cpu< funcType >(B, 420, B_length);

    for(int run = 0; run < ITERATIONS; run++){
        hipEvent_t start_event, stop_event;

        CUDA_RT_CALL(hipEventCreate(&start_event));
        CUDA_RT_CALL(hipEventCreate(&stop_event));

        CUDA_RT_CALL(hipEventRecord(start_event));
        e = multiGPU::MMM_streams< funcType, TILE >(A,B,C, HEIGHT_A, WIDTH_B, HEIGHT_B, streams);
        CUDA_RT_CALL(e);
        CUDA_RT_CALL(hipEventRecord(stop_event));
        CUDA_RT_CALL(hipEventSynchronize(stop_event));

        float ms;
        CUDA_RT_CALL(hipEventElapsedTime(&ms, start_event, stop_event));
        if(run != 0) output << ms << "\n";
    }

    hipFree(A);
    hipFree(B);
    hipFree(C_multi);
}