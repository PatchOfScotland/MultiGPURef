#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"

#define N 1e6


#define ENABLEPEERACCESS 1

typedef int funcType;

int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    funcType* A;
    float ms;
    hipEvent_t start, stop;

    for(int run = 0; run < ITERATIONS + 1; run++){
        CUDA_RT_CALL(hipEventCreate(&start));
        CUDA_RT_CALL(hipEventCreate(&stop));

        CUDA_RT_CALL(hipEventRecord(start));
        CUDA_RT_CALL(hipMallocManaged(&A, N*sizeof(funcType)));
        
        CUDA_RT_CALL(init_arr< funcType >(A, 1337, N));
        CUDA_RT_CALL(hipEventRecord(stop));
        CUDA_RT_CALL(hipEventSynchronize(stop));

        CUDA_RT_CALL(hipEventElapsedTime(&ms, start, stop));
        output << ms << "\n";

        hipFree(A);
        CUDA_RT_CALL(hipEventDestroy(start));
        CUDA_RT_CALL(hipEventDestroy(stop));

    }
    
    // may make this multicore?    


}