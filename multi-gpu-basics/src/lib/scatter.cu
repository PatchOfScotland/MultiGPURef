#include "hip/hip_runtime.h"
#ifndef SCATTER_H
#define SCATTER_H

#include "constants.cu.h"
#include "helpers.cu.h"

namespace singleGPU {
    template<class T>
    __global__ void scatter_kernel(T* data_old, long* idxs, T* data_in, size_t N_data, size_t N_idx){
        size_t gIdx = blockDim.x * blockIdx.x + threadIdx.x;
        if(gIdx < N_idx){
            long dIdx = idxs[gIdx];
            if(0 < dIdx && dIdx < N_data){
                data_old[dIdx] = data_in[gIdx];
            }
        }
    }

    template<class T>
    hipError_t scatter(void** args){
        T* data_old = *(T**)args[0];
        long* idxs = *(long**)args[1];
        T* data_in = *(T**)args[2];
        size_t N_data = *(size_t*)args[3];
        size_t N_idx = *(size_t*)args[4];

        const int64_t blockSize = 1024;
        const int64_t blocknum  = (N_idx + blockSize - 1) / blockSize;
        scatter_kernel< T ><<<blocknum, blockSize>>>(data_old, idxs, data_in, N_data, N_idx);
        return hipGetLastError();
    }
} // namespace singleGPU

namespace multiGPU {
    template<class T>
    __global__ void scatterUM_kernel(T* data_old, int64_t* idxs, T* data_in, size_t N_data, size_t N_idx, int devID){
        size_t gIdx = devID * blockDim.x * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
        if(gIdx < N_idx){
            long dIdx = idxs[gIdx];
            if(0 < dIdx && dIdx < N_data){
                data_old[dIdx] = data_in[gIdx];
            }
        }
    }

    template<class T>
    hipError_t scatterUM(void** args){
        T* data_old = *(T**)args[0];
        long* idxs = *(long**)args[1];
        T* data_in = *(T**)args[2];
        size_t N_data = *(size_t*)args[3];
        size_t N_idx = *(size_t*)args[4];

        int Device;
        hipGetDevice(&Device);
        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);
        const int64_t blockSize = 1024;
        const int64_t blockNum  = (N_idx + blockSize - 1) / blockSize;
        const int64_t blockPerDevice = blockNum / DeviceCount + 1;

        for(int devID = 0; devID < DeviceCount; devID++){
            hipSetDevice(devID);
            scatterUM_kernel< T ><<<blockPerDevice, blockSize>>>(data_old, idxs, data_in, N_data, N_idx, devID);  
        }
        hipSetDevice(Device);
        return hipGetLastError();
    }
} // namespace multiGPU


#endif