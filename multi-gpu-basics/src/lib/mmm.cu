#include "hip/hip_runtime.h"
#ifndef MMM_H
#define MMM_H

#include "constants.cu.h"
#include "helpers.cu.h"

namespace singleGPU {
    template <class ElTp, int T>
    __global__ void matMultRegTiledKernel(
        const ElTp* __restrict__ A,
        const ElTp* __restrict__ B,
        ElTp* C,
        const int heightA,
        const int widthB,
        const int widthA
    ) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA;
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  bidy * T;
        //int const bdimx = blockDim.x; // =Tile
        //int const bdimy = blockDim.y; // =Tile

        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + bidy * T < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b;
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
                __syncthreads();
            }

            for(int i = 0; i < T; i++){
                if ((ii + i) < heightA && j < widthB)  {
                    C[(i + ii)*widthB + j] = Creg[i];
                }
            }
        }
    }

    template< class ElTp, int T>
    hipError_t MMM( void** args){
            const ElTp* A = *(ElTp**)args[0];
            const ElTp* B = *(ElTp**)args[1];
            ElTp* C = *(ElTp**)args[2];
            const int A_height = *(int*)args[3];
            const int B_width  = *(int*)args[4];
            const int B_height = *(int*)args[5];

            dim3 block(T, T, 1);
            int grid_x = ceil((float)B_width / (T * T));
            int grid_y = ceil((float)A_height / (T));
            dim3 grid(grid_x, grid_y, 1);


            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A, B, C, A_height, B_width, B_height);
            return hipGetLastError();
    }
}

namespace multiGPU {
    template <class ElTp, int T>
    __global__ void matMultRegTiledKernel(
            const ElTp* __restrict__ A,
            const ElTp* __restrict__ B,
            ElTp* C,
            const int heightA,
            const int widthB,
            const int widthA,
            const int devID
        ) {
        __shared__ ElTp Ash[T][T];

        ElTp Creg[T];

        int const heightB = widthA;
        int const tidx = threadIdx.x;
        int const tidy = threadIdx.y;
        int const bidx = blockIdx.x;
        int const bidy = blockIdx.y;
        int const jjj = bidx * T * T;
        int const jj  = jjj + tidy * T;
        int const j   = jj + tidx;
        int const ii =  gridDim.y * T * devID + bidy * T;


        #pragma unroll
        for(int i = 0; i < T; i++) {
            Creg[i] = 0.0;
        }

        for(int kk = 0; kk < widthA; kk += T){
            //Copy A into temp memory
            if ( tidy + ii < heightA && kk + tidx < widthA ) {
                Ash[tidy][tidx] = A[(tidy + ii)*widthA + kk + tidx]; // Ash[tidy][tidx] = A[tidy + bidy * T][kk + tidx]
            } else {
                Ash[tidy][tidx] = 0.0;
            }
            __syncthreads();
            for(int k = 0; k < T; k++){
                //Copy B into a register
                float b;
                if ((k + kk) < heightB && j < widthB ) {
                    b = B[(k + kk) * widthB + j];
                } else {
                    b = 0.0;
                }

                #pragma unroll
                for(int i = 0; i < T; i++){
                    Creg[i] += Ash[i][k] * b;
                }
            }
            __syncthreads();
        }
        for(int i = 0; i < T; i++){
            if ((ii + i) < heightA && j < widthB)  {
                C[(ii + i) * widthB + j] = Creg[i];

            }
        }
    }

    template< class ElTp, int T>
    hipError_t MMM(void** args){
        // Extract Args
        const ElTp* A = *(ElTp**)args[0];
        const ElTp* B = *(ElTp**)args[1];
        ElTp* C = *(ElTp**)args[2];
        const int A_height = *(int*)args[3];
        const int B_width = *(int*)args[4];
        const int B_height = *(int*)args[5];


        int DeviceCount;
        hipGetDeviceCount(&DeviceCount);

        int Device = -1;
        hipGetDevice(&Device);

        const dim3 block(T, T, 1);
        const int grid_x_total = ceil((float)B_width / (T * T));
        const int grid_y_total = ceil((float)A_height / (T));

        const int grid_x = grid_x_total; // Keep this the same value and divide over the Y's
        const int grid_y = (grid_y_total + DeviceCount - 1) / DeviceCount; // Same trick to get matching blocksizes

        const dim3 grid(grid_x, grid_y, 1);

        for(int dev_id = 0; dev_id < DeviceCount; dev_id++){
            hipSetDevice(dev_id);
            matMultRegTiledKernel< ElTp, T ><<<grid, block>>>(A,B,C, A_height, B_width, B_height, dev_id);
        }

        hipSetDevice(Device);

        return hipGetLastError();
    }
}


#endif