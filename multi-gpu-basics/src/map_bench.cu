#include <iostream>
#include <fstream>
#include "constants.cu.h"
#include "helpers.cu.h"
#include "map.cu"

#define ENABLEPEERACCESS 1

typedef int funcType;


int main(int argc, char* argv[]){

    std::ofstream output;

    if (argc == 2){
        output.open(argv[1]);
    } else if (argc > 2) {
        std::cout << "Usage filename\n";
        exit(1);
    } else {
        output.open("/dev/null");
    }
    
    
    #if ENABLEPEERACCESS
    EnablePeerAccess();
    #endif

    int Device; 
    hipGetDevice(&Device);
    int Devices;
    hipGetDeviceCount(&Devices);
    hipStream_t streams[Devices];

    for(int devID = 0; devID < Devices; devID++){
        hipSetDevice(devID);
        hipStreamCreate(&streams[devID]);
    }

    hipError_t e;
    funcType* in;
    funcType* out;


    CUDA_RT_CALL(hipMallocManaged(&in, ARRAY_LENGTH*sizeof(funcType)));
    CUDA_RT_CALL(hipMallocManaged(&out, ARRAY_LENGTH*sizeof(funcType)));
    
    init_array_cpu< funcType >(in, 1337, ARRAY_LENGTH);
    

    for(int run = 0; run < ITERATIONS + 1; run++){
        hipEvent_t start_event, stop_event;

        CUDA_RT_CALL(hipEventCreate(&start_event));
        CUDA_RT_CALL(hipEventCreate(&stop_event));

        CUDA_RT_CALL(hipEventRecord(start_event));
        e = multiGPU::ApplyMap< MapBasic<funcType> >(in, out, ARRAY_LENGTH);
        CUDA_RT_CALL(e);
        CUDA_RT_CALL(hipDeviceSynchronize());
        CUDA_RT_CALL(hipEventRecord(stop_event));
        CUDA_RT_CALL(hipEventSynchronize(stop_event));

        float ms;
        CUDA_RT_CALL(hipEventElapsedTime(&ms, start_event, stop_event));
        if(run != 0) output << ms << "\n";
    }

    hipFree(in);
    hipFree(out);
}