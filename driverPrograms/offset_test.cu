#include <stdint.h>
#include <stddef.h>
#include <stdbool.h>
#include <stdio.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

#include "nvrtcHelpers.cu.h"
#include "constants.cu.h"


int main(int argc, char** argv){
  hipInit(0);

  hipDevice_t dev;
  hipCtx_t ctx;
  CUDA_SAFE_CALL(hipDeviceGet(&dev, 0));
  CUDA_SAFE_CALL(hipCtxCreate(&ctx, dev));

  int* arr = (int*)malloc(10*sizeof(int));
  for(int i = 0; i < 10; i++){
    arr[i] = i + 1;
  }
  hipDeviceptr_t mem_in, mem_out;

  CUDA_SAFE_CALL(hipMallocManaged(&mem_in, sizeof(int)*10, hipMemAttachGlobal));
  CUDA_SAFE_CALL(hipMallocManaged(&mem_out, sizeof(int), hipMemAttachGlobal));

  CUDA_SAFE_CALL(hipMemcpyHtoD(mem_in, arr, sizeof(int)*10));
  CUDA_SAFE_CALL(hipMemcpyDtoD(mem_out, mem_in + sizeof(int), sizeof(int)));
  CUDA_SAFE_CALL(hipMemcpyDtoH(arr, mem_out, sizeof(int)));

  fprintf(stderr, "%d\n", arr[0]);

  free(arr);
  CUDA_SAFE_CALL(hipFree(mem_in));
  CUDA_SAFE_CALL(hipFree(mem_out));

  return 0;
}