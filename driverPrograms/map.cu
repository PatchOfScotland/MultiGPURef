#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE // Avoid possible double-definition warning.
#define _GNU_SOURCE
#endif

#include <stdint.h>
#include <stddef.h>
#include <stdbool.h>
#include <stdio.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

#include "nvrtcHelpers.cu.h"
#include "constants.cu.h"

#define ARRAY_LENGTH 1e8
#define GPU_RUNS 25
#define DEBUG
#define UNIFIED 1


typedef int functype;

const char* program = "extern \"C\" __global__ void mapFunction(                  \n\
        int *mem_in,                                                              \n\
        int *mem_out,                                                             \n\
        const size_t arr,                                                         \n\
        const int offset ){                                                       \n\
    int idx = blockDim.x*gridDim.x*offset + blockDim.x * blockIdx.x + threadIdx.x;\n\
    if(idx < arr){                                                                \n\
        mem_out[idx] = mem_in[idx] + 1;                                           \n\
    }                                                                             \n\
}\n";


int main(int argc, char** argv){
    hipInit(0);
    int DeviceCount;
    hipGetDeviceCount(&DeviceCount);

    size_t N = ARRAY_LENGTH;
    const u_int BlockSize = 1024;
    const u_int NumBlocks = (N + BlockSize - 1) / BlockSize;
    const int BlocksPerDevice = NumBlocks / DeviceCount + 1;
    const size_t bufferSize = N*sizeof(int);

    
    hipModule_t* modules   = (hipModule_t*)malloc(sizeof(hipModule_t)*DeviceCount);
    hipDevice_t* devices   = (hipDevice_t*)malloc(sizeof(hipDevice_t)*DeviceCount);
    hipCtx_t* contexts = (hipCtx_t*)malloc(sizeof(hipCtx_t)*DeviceCount);
    hipStream_t* streams   = (hipStream_t*)malloc(sizeof(hipStream_t)*DeviceCount);
    hipEvent_t*  BenchmarkEvents = (hipEvent_t*)malloc(sizeof(hipEvent_t)*DeviceCount*2);
    hipFunction_t* Kernels = (hipFunction_t*)malloc(sizeof(hipFunction_t)); 

    for(int devID = 0; devID < DeviceCount; devID++){
        CUDA_SAFE_CALL(hipDeviceGet(&devices[devID], devID));
        CUDA_SAFE_CALL(hipCtxCreate(&contexts[devID], hipDeviceScheduleAuto, devices[devID])); // This Automaticly set the device
        CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
        CUDA_SAFE_CALL(hipStreamCreateWithFlags(&streams[devID], hipStreamDefault));
        CUDA_SAFE_CALL(hipEventCreateWithFlags(&BenchmarkEvents[devID*2] ,hipEventDefault)); // Start Event
        CUDA_SAFE_CALL(hipEventCreateWithFlags(&BenchmarkEvents[devID*2 + 1] ,hipEventDefault)); // Stop Event
    }

    char** functionNames = (char**)malloc(sizeof(char*));
    char* functionName_1 = "mapFunction";
    functionNames[0] = functionName_1;

    compileFunctions(program, functionNames, Kernels, 1, modules, contexts, DeviceCount);

    hipDeviceptr_t mem_in, mem_out;
    #if UNIFIED
    CUDA_SAFE_CALL(hipMallocManaged(&mem_in,  bufferSize, hipMemAttachGlobal));
    CUDA_SAFE_CALL(hipMallocManaged(&mem_out, bufferSize, hipMemAttachGlobal));
    #else
    CUDA_SAFE_CALL(hipMalloc(&mem_in, bufferSize));
    CUDA_SAFE_CALL(hipMalloc(&mem_out, bufferSize));
    #endif

    int* destData = (int*)malloc(N*sizeof(int));    

    #if UNIFIED
    int* hostData = (int*)mem_in; //You can do this, which looks horrible
    for(int i = 0; i < N; i++){
        hostData[i] = i;
    }
    #else
    int* hostData = (int*)malloc(N*sizeof(int));
    for(int i = 0; i < N; i++){
        hostData[i] = i;
    }
    CUDA_SAFE_CALL(hipMemcpyHtoD(mem_in, hostData, N*sizeof(int)));
    #endif


    #if UNIFIED
    for(int devID = 0; devID < DeviceCount; devID++){
        CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
        const size_t ElemsPerDevice = BlocksPerDevice*BlockSize;
        const size_t offset = ElemsPerDevice * devID;
        const size_t ElementsToPrefetch = (offset + ElemsPerDevice < N) ? ElemsPerDevice : N - offset;
        CUDA_SAFE_CALL(hipMemPrefetchAsync(mem_in + offset, ElementsToPrefetch*sizeof(int), devices[devID], streams[devID]));
    } 
    for(int devID = 0; devID < DeviceCount; devID++){
        CUDA_SAFE_CALL(hipStreamSynchronize(streams[devID]));
    }

    #endif


    for(int run = 0; run < GPU_RUNS; run++){
        
    for(int devID = 0; devID < DeviceCount; devID++){
            CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
            void *args[] = {&mem_in, &mem_out, &N, &devID};
            CUDA_SAFE_CALL(hipEventRecord(BenchmarkEvents[devID*2], streams[devID]));
            CUDA_SAFE_CALL(hipModuleLaunchKernel(Kernels[0], 
                BlocksPerDevice, 1, 1, 
                BlockSize, 1 ,1 , 
                0, streams[devID], 
                args, 0
            ));
            CUDA_SAFE_CALL(hipEventRecord(BenchmarkEvents[devID*2 + 1], streams[devID]));
        }
        float runTimes[DeviceCount];
        for(int devID = 0; devID < DeviceCount; devID++){
            CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
            CUDA_SAFE_CALL(hipStreamSynchronize(streams[devID]));
            CUDA_SAFE_CALL(hipEventElapsedTime(&runTimes[devID], BenchmarkEvents[devID*2], BenchmarkEvents[devID*2 + 1]));
            printf("%f\n", runTimes[devID]);
        }
    }

    CUDA_SAFE_CALL(hipMemcpyDtoH(destData, mem_out, N*sizeof(int)));

    for(int i = 0; i < N; i++){
        if(destData[i] != hostData[i] + 1){
            printf("Error at Index :%d\n", i);
        }
    }

    
    // Free data
    for(int devID = 0; devID < DeviceCount; devID++){
        CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
        hipModuleUnload(modules[devID]);
    }

    for(int devID = 0; devID < DeviceCount; devID++){
        CUDA_SAFE_CALL(hipCtxSetCurrent(contexts[devID]));
        CUDA_SAFE_CALL(hipEventDestroy(BenchmarkEvents[devID*2]));
        CUDA_SAFE_CALL(hipEventDestroy(BenchmarkEvents[devID*2 + 1]));
        CUDA_SAFE_CALL(hipStreamDestroy(streams[devID])); //Destroy Streams first
        CUDA_SAFE_CALL(hipCtxDestroy(contexts[devID]));
    }


    
    free(Kernels);
    free(functionNames);

    free(streams);
    free(contexts);
    free(devices);



}